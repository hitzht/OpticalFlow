
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <float.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <vector>
#include <chrono>

// #include "Matchcommon.h"
#define Radius 1
// #define NPixel 8
#define Deltat 0.00001
// int NBlocks;
// int Blocks;
// int binNum;

int find_option( int argc, char **argv, const char *option )
{
    for( int i = 1; i < argc; i++ )
        if( strcmp( argv[i], option ) == 0 )
            return i;
    return -1;
}

int read_int( int argc, char **argv, const char *option, int default_value ) // 
{
    int iplace = find_option( argc, argv, option ); 
    if( iplace >= 0 && iplace < argc-1 ) // 
        return atoi( argv[iplace+1] ); // Ascii to integer conversion
    return default_value;
}


double read_timer( )
{
    static bool initialized = false;
    static struct timeval start;
    struct timeval end;
    if( !initialized )
    {
        gettimeofday( &start, NULL );
        initialized = true;
    }
    gettimeofday( &end, NULL );
    return (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
}

//Initializing events
void init_events( int n, int *g )
{
    srand48( time( NULL ) );

    for (int i = 0; i <  n; i++) 
        for (int j = 0; j < n; j++) 
            g[i*n+j] = drand48()+.5;
}

__device__ void HammingDistance(int *tmd, int *tm2d, int tmdaddr, int tm2daddr, int *temp, int N, int NPixel)
{
            int xtid = threadIdx.x + blockIdx.x * blockDim.x;
            int ytid = threadIdx.y + blockIdx.y * blockDim.y;
            int xoffset = gridDim.x*blockDim.x;
            int yoffset = gridDim.y*blockDim.y;
         for (int k = xtid; k < NPixel; k+=xoffset)
                for (int l = ytid; l < NPixel; l+=yoffset)
                    *temp += int (tmd[tmdaddr+(k*N+l)] ^ tm2d[tm2daddr+(k*N+l)]);
}

__device__ void OpticalFlow(int *tmd, int *tm2d, int tmdaddr, int tm2daddr, int *velocity, int N, int NPixel)
{
        int xtid = threadIdx.x + blockIdx.x * blockDim.x;
        int ytid = threadIdx.y + blockIdx.y * blockDim.y;
        int xoffset = gridDim.x*blockDim.x;
        int yoffset = gridDim.y*blockDim.y;
         for (int k = xtid; k < NPixel; k+=xoffset)
                for (int l = ytid; l < NPixel; l+=yoffset)                    
                    velocity[tmdaddr+(k*N+l)] = int (tmd[tmdaddr+(k*N+l)] - tm2d[tm2daddr+(k*N+l)]);
}

__global__ void TargetDefining( int* tmd, int* tm2d, int* Targetind, int* targetvalue, int N, int NPixel)
{
    // // NBlocks = int (N/NPixel);
        int xtid = threadIdx.x + blockIdx.x * blockDim.x;
        int ytid = threadIdx.y + blockIdx.y * blockDim.y;
        int xoffset = gridDim.x*blockDim.x;
        int yoffset = gridDim.y*blockDim.y;
        for (int i = xtid; i < (N/NPixel); i+=xoffset){
            for (int j = ytid; j < (N/NPixel); j+=yoffset){
            Targetind[i*(N/NPixel) + j] = i*NPixel*N + j*NPixel; // Assign block in t-2d as target
            HammingDistance(tmd, tm2d, i*NPixel*N + j*NPixel, i*NPixel*N + j*NPixel, &targetvalue[i*(N/NPixel) + j], N,NPixel );
            }
        }
}

__constant__ const int dir[8][2]={{-1,-1},{0,-1},{1,-1},{1,0},{1,1},{0,1},{-1,1},{-1,0}};
__global__ void OpticalMatching(int* __restrict__ tmd, int* __restrict__ tm2d, int* __restrict__ Targetind, int* __restrict__ targetvalue, int N, int NPixel)
{
        int xtid = threadIdx.x + blockIdx.x * blockDim.x;
        int ytid = threadIdx.y + blockIdx.y * blockDim.y;
        int xoffset = gridDim.x*blockDim.x;
        int yoffset = gridDim.y*blockDim.y;
        for (int i = xtid; i <  N/NPixel; i+=xoffset){
            for (int j = ytid; j <  N/NPixel; j+=yoffset){
                for(int t=0;t<8;t++){
                              int x = (i + dir[t][0]);
                              int y = (j + dir[t][1]);
                        if (x*NPixel >= 0 && x*NPixel < N && y*NPixel >= 0 && y*NPixel < N)
                        {
                            int temp = 0;
                            // Calculating the hamming distance
                            HammingDistance(tmd, tm2d, i*NPixel*N + j*NPixel, (x*NPixel*N) + (y*NPixel), &temp, N, NPixel);
                            // printf("temp for block %d where dx is %d and dy is %d is %d\n",i* N/NPixel+j,dx,dy,temp);
                            // for (int k = 0; k < NPixel; k++)
                            //     for (int l = 0; l < NPixel; l++)
                            //         temp = temp + int (tmd[(i+k)*N+(j+l)] xor tm2d[(i+k+dx*Radius)*N+(j+l+dy*Radius)]);
                            // printf("temp is %d\n",temp);
                            if (temp < targetvalue[i* N/NPixel + j])
                            {
                                targetvalue[i* N/NPixel + j] = temp;
                                // printf("targetvalue[%d] is %d\n",i* N/NPixel+j,targetvalue[i* N/NPixel + j]);
                                Targetind[i* N/NPixel + j] = x*NPixel*N + y*NPixel;
                            }
                        }
                }
            }
        }
}

__global__ void OpticalFlowCalculation(int *tmd, int *tm2d, int *Targetind, int *velocity, int N, int NPixel){
        int xtid = threadIdx.x + blockIdx.x * blockDim.x;
        int ytid = threadIdx.y + blockIdx.y * blockDim.y;
        int xoffset = gridDim.x*blockDim.x;
        int yoffset = gridDim.y*blockDim.y;
        for (int i = xtid; i < (N/NPixel); i+=xoffset){
            for (int j = ytid; j < (N/NPixel); j+=yoffset){
                OpticalFlow(tmd, tm2d, i*NPixel*N + j*NPixel, Targetind[i*(N/NPixel)+j], velocity, N, NPixel);
            }
        }
}

int main(int argc, char **argv)
{
int N = read_int( argc, argv, "-n", 64 ); // the number of particles
int NUM_THREADS = read_int( argc, argv, "-t", 256 ); // the number of particles
int blks = read_int( argc, argv, "-b", 1024 ); // the number of particles
int NPixel = read_int( argc, argv, "-p", 8 ); // the number of particles


int NBlocks = int (N/NPixel);
// binNum = int(N / NPixel); // Should be around sqrt(N/2)

int *tmd = (int*) malloc( N * N * sizeof(int) );
int *tm2d = (int*) malloc( N * N * sizeof(int) );
int *velocity = (int*) malloc( N * N * sizeof(int) );

int *Targetind = (int*) malloc( NBlocks * NBlocks * sizeof(int) );
int *targetvalue = (int*) malloc( NBlocks * NBlocks * sizeof(int) );

int * d_tmd,*d_tm2d, *d_velocity, *d_Targetind, *d_targetvalue;
hipMalloc((void **) &d_tmd, N * N * sizeof(int));
hipMalloc((void **) &d_tm2d, N * N * sizeof(int));
hipMalloc((void **) &d_velocity, N * N * sizeof(int));
hipMalloc((void **) &d_Targetind, NBlocks * NBlocks * sizeof(int));
hipMalloc((void **) &d_targetvalue, NBlocks * NBlocks * sizeof(int));

init_events(N, tmd);
init_events(N, tm2d);

hipDeviceSynchronize();
double copy_time = read_timer( );
hipMemcpy(d_tmd, tmd, N * N * sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_tm2d, tm2d, N * N * sizeof(int), hipMemcpyHostToDevice);
hipDeviceSynchronize();
copy_time = read_timer( ) - copy_time;

auto begin_sim = std::chrono::high_resolution_clock::now();
////////////////////////////////////////////////////////////////////////////////     
////////////////////////////  Target Defining      ////////////////////////////
////////////////////////////////////////////////////////////////////////////////
        int threadNum = NUM_THREADS;
        // int blks = min(1024,(N*N + NUM_THREADS - 1) / NUM_THREADS);
        int blockNum = blks;//min(512,(n+threadNum-1)/threadNum);

        hipMemset(d_Targetind, 0, NBlocks * NBlocks * sizeof(int));
        hipMemset(d_targetvalue, 0, NBlocks * NBlocks * sizeof(int));
        TargetDefining<<<blockNum,threadNum>>>(d_tmd, d_tm2d, d_Targetind, d_targetvalue, N, NPixel );
        // cudaMemcpy(Targetind, d_Targetind, NBlocks * NBlocks * sizeof(int), cudaMemcpyDeviceToHost);
        // cudaMemcpy(targetvalue, d_targetvalue, NBlocks * NBlocks * sizeof(int), cudaMemcpyDeviceToHost);

        // for (int i = 0; i < N; i+=NPixel)
        // {
        //     for (int j = 0; j < N; j+=NPixel)
        //     {
        //     count = ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel;
        //     Targetind[count] = i*N + j; // Assign block in t-2d as target
        //     // printf("count= %d, ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel= %d\n", count, ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel);
        //     HammingDistance(NPixel, N, i*N + j, i*N + j, tmd, tm2d, &targetvalue[count]);
        //     }
        // }

////////////////////////////////////////////////////////////////////////////////     
////////////////////////////  Optical Matching      ////////////////////////////
////////////////////////////////////////////////////////////////////////////////

        // cudaMemcpy(d_Targetind, Targetind, NBlocks * NBlocks * sizeof(int), cudaMemcpyHostToDevice);
        // cudaMemcpy(d_targetvalue, targetvalue, NBlocks * NBlocks * sizeof(int), cudaMemcpyHostToDevice);
        OpticalMatching<<<blockNum,threadNum>>>(d_tmd, d_tm2d, d_Targetind, d_targetvalue, N, NPixel);
        // cudaMemcpy(Targetind, d_Targetind, NBlocks * NBlocks * sizeof(int), cudaMemcpyDeviceToHost);

        //  for (int i = 0; i < N; i+=NPixel)
        // {
        //     for (int j = 0; j < N; j+=NPixel)
        //     {
        //         count = ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel;
        //         // int *temp = (int*) calloc((2*Radius +1)*(2*Radius +1), sizeof(int));
        //         // Targetind[i*NBlocks+j] = (i * NBlocks + j);
        //         for (int dx = -Radius; dx <= Radius; dx++)   //Search over nearby 8 blocks and the target block 
        //         {
        //             for (int dy = -Radius; dy <= Radius; dy++)
        //             {
        //                 if (i + dx*NPixel >= 0 && i + dx*NPixel < N && j + dy*NPixel >= 0 && j + dy*NPixel < N)
        //                 {
        //                     temp = 0;
        //                     // Calculating the hamming distance
        //                     HammingDistance(NPixel, N, i*N + j, (i+dx*Radius)*N + (j+dy*Radius), tmd, tm2d, &temp);
        //                     // for (int k = 0; k < NPixel; k++)
        //                     //     for (int l = 0; l < NPixel; l++)
        //                     //         temp = temp + int (tmd[(i+k)*N+(j+l)] xor tm2d[(i+k+dx*Radius)*N+(j+l+dy*Radius)]);
        //                     // printf("temp is %d\n",temp);
        //                     if (temp < targetvalue[count])
        //                     {
        //                     	targetvalue[count] = temp;
        //                         Targetind[count] = (i + dx*NPixel)*N + (j + dy*NPixel);
        //                         // int Targetind[count] = 0;
        //                     }
        //                 }
        //             }
        //         }
        //     }
        // }
///////////////////////////////////////////////////////////////////////////////////     
////////////////////////////////////  Optical Flow Calculation ////////////////////
///////////////////////////////////////////////////////////////////////////////////
        // cudaMemcpy(d_Targetind, Targetind, NBlocks * NBlocks * sizeof(int), cudaMemcpyHostToDevice);
        hipMemset(d_velocity, 0, N * N * sizeof(int) );
        OpticalFlowCalculation<<<blockNum,threadNum>>>(d_tmd, d_tm2d, d_Targetind, d_velocity, N, NPixel);
        hipMemcpy(velocity, d_velocity, N * N * sizeof(int), hipMemcpyDeviceToHost);

//         for (int i = 0; i < N; i+=NPixel)
//         {
//             for (int j = 0; j < N; j+=NPixel)
//             {
//                 count = ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel;
// // Calculating the Optical Flow
//                 OpticalFlow(NPixel, N, i*N + j, Targetind[count], tmd, tm2d, velocity);                
//                 // for (int k = 0; k < NPixel; k++)
//                 //     for (int l = 0; l < NPixel; l++)
//                 //         OpticalFlow[(i+k)*N+(j+l)] = tmd[(i+k)*N+(j+l)] - tm2d[Targetind[count]+(k*N+l)];
//             }
//         }

   hipDeviceSynchronize();
  auto end_sim = std::chrono::high_resolution_clock::now();
  double simdur = std::chrono::duration <double> (end_sim - begin_sim).count();
  printf( "N: %d, NPixels: %d, NBlocks: %d, Radius: %d, Threads: %d, Blocks: %d, GPGPU ST: %g, CT: %g\n", N, NPixel, NBlocks, Radius, NUM_THREADS, blockNum, simdur, copy_time);


// int i, j, count = 0; 
// int *B[N];
//     for (i=0; i<M; i++)
//          B[i] = (int *)malloc(N * sizeof(int));

    // for (int i = 0; i <  N; i++) 
    //   for (int j = 0; j < N; j++)
    //      printf("tmd in position %d*%d is:%d\n",i, j, tmd[i*N+j]);
hipFree(d_tmd);
hipFree(d_tm2d);
hipFree(d_velocity);

free( tmd );
free( tm2d );
free( velocity );
free( Targetind );
free( targetvalue );
return 0;
}
// //module load cuda
// //salloc -N 1 -t 01:30:00 -p gpu

// #include <stdlib.h>
// #include <stdio.h>
// #include <assert.h>
// #include <float.h>
// #include <string.h>
// #include <math.h>
// #include <time.h>
// #include <sys/time.h>
// #include <vector>
// // #include "Matchcommon.h"
// #define NUM_THREADS 256
// #define Radius 1
// #define NPixel 8
// #define Deltat 0.00001
// int NBlocks;
// int Blocks;
// int binNum;

// int find_option( int argc, char **argv, const char *option )
// {
//     for( int i = 1; i < argc; i++ )
//         if( strcmp( argv[i], option ) == 0 )
//             return i;
//     return -1;
// }

// int read_int( int argc, char **argv, const char *option, int default_value ) // 
// {
//     int iplace = find_option( argc, argv, option ); 
//     if( iplace >= 0 && iplace < argc-1 ) // 
//         return atoi( argv[iplace+1] ); // Ascii to integer conversion
//     return default_value;
// }


// double read_timer( )
// {
//     static bool initialized = false;
//     static struct timeval start;
//     struct timeval end;
//     if( !initialized )
//     {
//         gettimeofday( &start, NULL );
//         initialized = true;
//     }
//     gettimeofday( &end, NULL );
//     return (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
// }

// void init_events( int n, int *g1 , int *g2)
// {
//     srand48( time( NULL ) );

//     for (int i = 0; i <  n; i++) {
//         for (int j = 0; j < n; j++){ 
//             g1[i*n+j] = drand48()+.5;
//             g2[i*n+j] = drand48()+.5;
//         }
//     }
// }

// __device__ void HammingDistance(int *tmd, int *tm2d, int tmdaddr, int tm2daddr, int *temp, int N )
// {
//          //    int xtid = threadIdx.x + blockIdx.x * blockDim.x;
//          //    int ytid = threadIdx.y + blockIdx.y * blockDim.y;
//          //    int xoffset = gridDim.x*blockDim.x;
//          //    int yoffset = gridDim.y*blockDim.y;
//          // for (int k = xtid; k < NPixel; k+=xoffset){
//          //        for (int l = ytid; l < NPixel; l+=yoffset){
//          for (int k = 0; k < NPixel; k++){
//                 for (int l = 0; l < NPixel; l++){
//                     *temp += int (tmd[tmdaddr+(k*N+l)] ^ tm2d[tm2daddr+(k*N+l)]);
//                     // atomicAdd(temp,int (tmd[tmdaddr+(k*N+l)] ^ tm2d[tm2daddr+(k*N+l)]));
//         // printf("tm2daddr is %d+ (k*N+l) is %d and tm2daddr+(k*N+l) is %d and temp is %d\n", tm2daddr, (k*N+l),tm2daddr+(k*N+l),*temp);
//                 }
//          }
// }

// __device__ void OpticalFlow(int *tmd, int *tm2d, int tmdaddr, int tm2daddr, int *velocity, int N )
// {
//         // int xtid = threadIdx.x + blockIdx.x * blockDim.x;
//         // int ytid = threadIdx.y + blockIdx.y * blockDim.y;
//         // int xoffset = gridDim.x*blockDim.x;
//         // int yoffset = gridDim.y*blockDim.y;
//         //  for (int k = xtid; k < NPixel; k+=xoffset)
//         //         for (int l = ytid; l < NPixel; l+=yoffset)                    
//          for (int k = 0; k < NPixel; k++){
//                 for (int l = 0; l < NPixel; l++){
//                     velocity[tmdaddr+(k*N+l)] = int (tmd[tmdaddr+(k*N+l)] - tm2d[tm2daddr+(k*N+l)]);
//                 }
//             }
// }

// __global__ void TargetDefining( int* tmd, int* tm2d, int* Targetind, int* targetvalue, int N )
// {
//     // // NBlocks = int (N/NPixel);
//         int xtid = threadIdx.x + blockIdx.x * blockDim.x;
//         int ytid = threadIdx.y + blockIdx.y * blockDim.y;
//         int xoffset = gridDim.x*blockDim.x;
//         int yoffset = gridDim.y*blockDim.y;
//         for (int i = xtid; i < (N/NPixel); i+=xoffset){
//             for (int j = ytid; j < (N/NPixel); j+=yoffset){
//             Targetind[i*(N/NPixel) + j] = i*(N/NPixel) + j; // Assign block in t-2d as target
//             HammingDistance(tmd, tm2d, i*NPixel*N + j*NPixel, i*NPixel*N + j*NPixel, &targetvalue[i*(N/NPixel) + j], N );
//             // printf("targetvalue[i*NBlocks + j] is %d\n",targetvalue[i*(N/NPixel) + j]);
//             }
//         }
// }

// __constant__ const int dir[8][2]={{-1,-1},{0,-1},{1,-1},{1,0},{1,1},{0,1},{-1,1},{-1,0}};

// __global__ void OpticalMatching(int* __restrict__ tmd, int* __restrict__ tm2d, int* __restrict__ Targetind, int* __restrict__ targetvalue, int N){
//         int xtid = threadIdx.x + blockIdx.x * blockDim.x;
//         int ytid = threadIdx.y + blockIdx.y * blockDim.y;
//         int xoffset = gridDim.x*blockDim.x;
//         int yoffset = gridDim.y*blockDim.y;
//         for (int i = xtid; i <  N/NPixel; i+=xoffset){
//             for (int j = ytid; j <  N/NPixel; j+=yoffset){
//                 for(int t=0;t<8;t++){
//                               int x = (i + dir[t][0]);
//                               int y = (j + dir[t][1]);
//                 // for (int dx = -Radius; dx <= Radius; dx++)   //Search over nearby 8 blocks and the target block 
//                 // {
//                 //     for (int dy = -Radius; dy <= Radius; dy++)
//                 //     {
//                         if (x*NPixel >= 0 && x*NPixel < N && y*NPixel >= 0 && y*NPixel < N)
//                         {
//                             int temp = 0;
//                             // Calculating the hamming distance
//                             HammingDistance(tmd, tm2d, i*NPixel*N + j*NPixel, ((i+dx)*NPixel*N) + ((j+dy)*NPixel), &temp, N);
//                             // printf("temp for block %d where dx is %d and dy is %d is %d\n",i* N/NPixel+j,dx,dy,temp);
//                             // for (int k = 0; k < NPixel; k++)
//                             //     for (int l = 0; l < NPixel; l++)
//                             //         temp = temp + int (tmd[(i+k)*N+(j+l)] xor tm2d[(i+k+dx*Radius)*N+(j+l+dy*Radius)]);
//                             // printf("temp is %d\n",temp);
//                             if (temp < targetvalue[i* N/NPixel + j])
//                             {
//                                 targetvalue[i* N/NPixel + j] = temp;
//                                 // printf("targetvalue[%d] is %d\n",i* N/NPixel+j,targetvalue[i* N/NPixel + j]);
//                                 Targetind[i* N/NPixel + j] = x* N/NPixel + y;
//                             }
//                         }
//                 }
//             }
//         }
// }

// __global__ void OpticalFlowCalculation(int *tmd, int *tm2d, int *Targetind, int *velocity, int N){
//         int xtid = threadIdx.x + blockIdx.x * blockDim.x;
//         int ytid = threadIdx.y + blockIdx.y * blockDim.y;
//         int xoffset = gridDim.x*blockDim.x;
//         int yoffset = gridDim.y*blockDim.y;
//         for (int i = xtid; i < (N/NPixel); i+=xoffset){
//             for (int j = ytid; j < (N/NPixel); j+=yoffset){
//                 int tm2daddri= (Targetind[i*(N/NPixel)+j]*NPixel)/N;
//                 int tm2daddrj= Targetind[i*(N/NPixel) + j]-tm2daddri*(N/NPixel);
//                 int tm2daddr= tm2daddri*NPixel*N+tm2daddrj*NPixel;
//                 OpticalFlow(tmd, tm2d, i*NPixel*N + j*NPixel, tm2daddr, velocity, N);
//             }
//         }
// }



// int main(int argc, char **argv)
// {
// int N = read_int( argc, argv, "-n", 32 ); // the number of particles
// NBlocks = int (N/NPixel);
// // binNum = int(N / NPixel); // Should be around sqrt(N/2)

// int *tmd = (int*) malloc( N * N * sizeof(int) );
// int *tm2d = (int*) malloc( N * N * sizeof(int) );
// int *velocity = (int*) malloc( N * N * sizeof(int) );

// int *Targetind = (int*) malloc( NBlocks * NBlocks * sizeof(int) );
// int *targetvalue = (int*) malloc( NBlocks * NBlocks * sizeof(int) );

// int * d_tmd,*d_tm2d, *d_velocity, *d_Targetind, *d_targetvalue;
// cudaMalloc((void **) &d_tmd, N * N * sizeof(int));
// cudaMalloc((void **) &d_tm2d, N * N * sizeof(int));
// cudaMalloc((void **) &d_velocity, N * N * sizeof(int));
// cudaMalloc((void **) &d_Targetind, NBlocks * NBlocks * sizeof(int));
// cudaMalloc((void **) &d_targetvalue, NBlocks * NBlocks * sizeof(int));

// init_events(N, tmd,tm2d);
// // /////////////////////////////////////////////////
// // ///////////////// just for test /////////////////
// // /////////////////////////////////////////////////
// //         printf("tmd is a 4*4 matrix in 2*2 block:\n");
// //         printf("|%d  %d | %d  %d|:\n",tmd[0],tmd[1],tmd[2],tmd[3]);
// //         printf("|%d  %d | %d  %d|:\n",tmd[4],tmd[5],tmd[6],tmd[7]);
// //         printf("---------------\n");
// //         printf("|%d  %d | %d  %d|:\n",tmd[8],tmd[9],tmd[10],tmd[11]);
// //         printf("|%d  %d | %d  %d|:\n",tmd[12],tmd[13],tmd[14],tmd[15]);

// //         printf("tm2d is a 4*4 matrix in 2*2 block:\n");
// //         printf("|%d  %d | %d  %d|:\n",tm2d[0],tm2d[1],tm2d[2],tm2d[3]);
// //         printf("|%d  %d | %d  %d|:\n",tm2d[4],tm2d[5],tm2d[6],tm2d[7]);
// //         printf("---------------\n");
// //         printf("|%d  %d | %d  %d|:\n",tm2d[8],tm2d[9],tm2d[10],tm2d[11]);
// //         printf("|%d  %d | %d  %d|:\n",tm2d[12],tm2d[13],tm2d[14],tm2d[15]);
// // /////////////////////////////////////////////////
// // /////////////////////////////////////////////////
// // /////////////////////////////////////////////////

// cudaDeviceSynchronize();
// double copy_time = read_timer();
// cudaMemcpy(d_tmd, tmd, N * N * sizeof(int), cudaMemcpyHostToDevice);
// cudaMemcpy(d_tm2d, tm2d, N * N * sizeof(int), cudaMemcpyHostToDevice);
// cudaDeviceSynchronize();
// copy_time = read_timer() - copy_time;

// double simulation_time = read_timer( );
// ////////////////////////////////////////////////////////////////////////////////     
// ////////////////////////////  Target Defining      ////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////
//         int threadNum = NUM_THREADS;
//         int blks = min(1024,(N*N + NUM_THREADS - 1) / NUM_THREADS);
//         int blockNum = blks;//min(512,(n+threadNum-1)/threadNum);

//         cudaMemset(d_Targetind, 0, NBlocks * NBlocks * sizeof(int));
//         cudaMemset(d_targetvalue, 0, NBlocks * NBlocks * sizeof(int));
//         TargetDefining<<<blockNum,threadNum>>>(d_tmd, d_tm2d, d_Targetind, d_targetvalue, N );
//         cudaMemcpy(Targetind, d_Targetind, NBlocks * NBlocks * sizeof(int), cudaMemcpyDeviceToHost);
//         cudaMemcpy(targetvalue, d_targetvalue, NBlocks * NBlocks * sizeof(int), cudaMemcpyDeviceToHost);

// // /////////////////////////////////////////////////
// // ///////////////// just for test /////////////////
// // /////////////////////////////////////////////////
// //         printf("Target Indices for 2*2 blocks:\n");
// //         printf("|%d | %d|:\n",Targetind[0],Targetind[1]);
// //         printf("|%d | %d|:\n",Targetind[2],Targetind[3]);
// //         printf("Target Values for 2*2 blocks:\n");
// //         printf("|%d | %d|:\n",targetvalue[0],targetvalue[1]);
// //         printf("|%d | %d|:\n",targetvalue[2],targetvalue[3]);
// // /////////////////////////////////////////////////
// // /////////////////////////////////////////////////
// // /////////////////////////////////////////////////

//         // for (int i = 0; i < N; i+=NPixel)
//         // {
//         //     for (int j = 0; j < N; j+=NPixel)
//         //     {
//         //     count = ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel;
//         //     Targetind[count] = i*N + j; // Assign block in t-2d as target
//         //     // printf("count= %d, ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel= %d\n", count, ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel);
//         //     HammingDistance(NPixel, N, i*N + j, i*N + j, tmd, tm2d, &targetvalue[count]);
//         //     }
//         // }

// ////////////////////////////////////////////////////////////////////////////////     
// ////////////////////////////  Optical Matching      ////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////
//         cudaMemcpy(d_Targetind, Targetind, NBlocks * NBlocks * sizeof(int), cudaMemcpyHostToDevice);
//         cudaMemcpy(d_targetvalue, targetvalue, NBlocks * NBlocks * sizeof(int), cudaMemcpyHostToDevice);
//         OpticalMatching<<<blockNum,threadNum>>>(d_tmd, d_tm2d, d_Targetind, d_targetvalue, N);
//         cudaMemcpy(Targetind, d_Targetind, NBlocks * NBlocks * sizeof(int), cudaMemcpyDeviceToHost);
//         // cudaMemcpy(targetvalue, d_targetvalue, NBlocks * NBlocks * sizeof(int), cudaMemcpyDeviceToHost);
// // /////////////////////////////////////////////////
// // ///////////////// just for test /////////////////
// // /////////////////////////////////////////////////
// //         printf("New Target Indices for 2*2 blocks:\n");
// //         printf("|%d | %d|:\n",Targetind[0],Targetind[1]);
// //         printf("|%d | %d|:\n",Targetind[2],Targetind[3]);
// //         printf("New Target Values for 2*2 blocks:\n");
// //         printf("|%d | %d|:\n",targetvalue[0],targetvalue[1]);
// //         printf("|%d | %d|:\n",targetvalue[2],targetvalue[3]);
// // /////////////////////////////////////////////////
// // /////////////////////////////////////////////////
// // /////////////////////////////////////////////////

//         //  for (int i = 0; i < N; i+=NPixel)
//         // {
//         //     for (int j = 0; j < N; j+=NPixel)
//         //     {
//         //         count = ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel;
//         //         // int *temp = (int*) calloc((2*Radius +1)*(2*Radius +1), sizeof(int));
//         //         // Targetind[i*NBlocks+j] = (i * NBlocks + j);
//         //         for (int dx = -Radius; dx <= Radius; dx++)   //Search over nearby 8 blocks and the target block 
//         //         {
//         //             for (int dy = -Radius; dy <= Radius; dy++)
//         //             {
//         //                 if (i + dx*NPixel >= 0 && i + dx*NPixel < N && j + dy*NPixel >= 0 && j + dy*NPixel < N)
//         //                 {
//         //                     temp = 0;
//         //                     // Calculating the hamming distance
//         //                     HammingDistance(NPixel, N, i*N + j, (i+dx*Radius)*N + (j+dy*Radius), tmd, tm2d, &temp);
//         //                     // for (int k = 0; k < NPixel; k++)
//         //                     //     for (int l = 0; l < NPixel; l++)
//         //                     //         temp = temp + int (tmd[(i+k)*N+(j+l)] xor tm2d[(i+k+dx*Radius)*N+(j+l+dy*Radius)]);
//         //                     // printf("temp is %d\n",temp);
//         //                     if (temp < targetvalue[count])
//         //                     {
//         //                      targetvalue[count] = temp;
//         //                         Targetind[count] = (i + dx*NPixel)*N + (j + dy*NPixel);
//         //                         // int Targetind[count] = 0;
//         //                     }
//         //                 }
//         //             }
//         //         }
//         //     }
//         // }
// ///////////////////////////////////////////////////////////////////////////////////     
// ////////////////////////////////////  Optical Flow Calculation ////////////////////
// ///////////////////////////////////////////////////////////////////////////////////
//         cudaMemcpy(d_Targetind, Targetind, NBlocks * NBlocks * sizeof(int), cudaMemcpyHostToDevice);
//         cudaMemset(d_velocity, 0, N * N * sizeof(int) );
//         OpticalFlowCalculation<<<blockNum,threadNum>>>(d_tmd, d_tm2d, d_Targetind, d_velocity, N);
//         // cudaMemcpy(velocity, d_velocity, N * N * sizeof(int), cudaMemcpyDeviceToHost);
// // /////////////////////////////////////////////////
// // ///////////////// just for test /////////////////
// // /////////////////////////////////////////////////
// //         printf("Diff for 4*4 matrix and 2*2 blocks:\n");
// //         printf("|%d  %d | %d  %d|:\n",velocity[0],velocity[1],velocity[2],velocity[3]);
// //         printf("|%d  %d | %d  %d|:\n",velocity[4],velocity[5],velocity[6],velocity[7]);
// //         printf("---------------\n");
// //         printf("|%d  %d | %d  %d|:\n",velocity[8],velocity[9],velocity[10],velocity[11]);
// //         printf("|%d  %d | %d  %d|:\n",velocity[12],velocity[13],velocity[14],velocity[15]);
// // /////////////////////////////////////////////////
// // /////////////////////////////////////////////////
// // /////////////////////////////////////////////////
// //         for (int i = 0; i < N; i+=NPixel)
// //         {
// //             for (int j = 0; j < N; j+=NPixel)
// //             {
// //                 count = ((i+NPixel-1)/NPixel)*NBlocks + (j+NPixel-1)/NPixel;
// // // Calculating the Optical Flow
// //                 OpticalFlow(NPixel, N, i*N + j, Targetind[count], tmd, tm2d, velocity);                
// //                 // for (int k = 0; k < NPixel; k++)
// //                 //     for (int l = 0; l < NPixel; l++)
// //                 //         OpticalFlow[(i+k)*N+(j+l)] = tmd[(i+k)*N+(j+l)] - tm2d[Targetind[count]+(k*N+l)];
// //             }
// //         }
//    cudaDeviceSynchronize();
//    simulation_time = read_timer( ) - simulation_time;
//    printf( "N: %d, NPixels: %d, NBlocks: %d, Radius: %d, GPGPU ST: %g s, CT: %g s\n", N, NPixel, N/NPixel, Radius, simulation_time,copy_time);


// // int i, j, count = 0; 
// // int *B[N];
// //     for (i=0; i<M; i++)
// //          B[i] = (int *)malloc(N * sizeof(int));

//     // for (int i = 0; i <  N; i++) 
//     //   for (int j = 0; j < N; j++)
//     //      printf("tmd in position %d*%d is:%d\n",i, j, tmd[i*N+j]);
// cudaFree(d_tmd);
// cudaFree(d_tm2d);
// cudaFree(d_velocity);

// free( tmd );
// free( tm2d );
// free( velocity );
// free( Targetind );
// free( targetvalue );
// return 0;
// }
// //module load cuda
// //salloc -N 1 -t 01:30:00 -p gpu

